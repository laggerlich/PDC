#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;

#define PI 3.14159265 

#define CSC(call)  \
do { \
	hipError_t state = call; \
	if (state != hipSuccess) { \
		fprintf(stderr, "ERROR: %s:%d. Message: %s\n", __FILE__,__LINE__,hipGetErrorString(state)); \
		exit(0); \
	} \
} while (0); \

__global__ void kernel(double* v1, long long n) {
    int i, idx = blockDim.x * blockIdx.x + threadIdx.x;
    long long offset = blockDim.x * gridDim.x;
    for (i = idx; i < n; i += offset) {
        v1[i] = sin(PI*((float)i/36));
    }
}

int main()
{
    long long n = 1000000;
    double* sin = (double*)malloc(n * sizeof(double));
    double* sin_dev = (double*)malloc(n * sizeof(double));

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    CSC(hipMalloc(&sin_dev, sizeof(double) * n));
    CSC(hipMemcpy(sin_dev, sin, sizeof(double) * n, hipMemcpyHostToDevice));

    hipEventRecord(start, 0);

    kernel << <256, 256 >> > (sin_dev, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    CSC(hipMemcpy(sin, sin_dev, sizeof(double) * n, hipMemcpyDeviceToHost));
    CSC(hipFree(sin_dev));

    for (long long i = 0; i < n; i++) {
        printf("%.3f\n", sin[i]);
    }

    printf("\n");
    free(sin);
    hipEventElapsedTime(&time, start, stop);
    printf("Time for the kernel: %f ms\n", time);
    return 0;
}
